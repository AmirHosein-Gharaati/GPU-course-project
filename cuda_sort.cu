#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

typedef struct mergeSortResult
{
    hipError_t cudaStatus;
    char *msg;
} mergeSortResult_t;

__global__ void mergeSortKernel(int *arr, int *aux, unsigned int blockSize, const unsigned int last)
{
    int x = threadIdx.x;
    int start = blockSize * x;
    int end = start + blockSize - 1;
    int mid = start + (blockSize / 2) - 1;
    int l = start, r = mid + 1, i = start;

    if (end > last)
    {
        end = last;
    }
    if (start == end || end <= mid)
    {
        return;
    }

    while (l <= mid && r <= end)
    {
        if (arr[l] <= arr[r])
        {
            aux[i++] = arr[l++];
        }
        else
        {
            aux[i++] = arr[r++];
        }
    }

    while (l <= mid)
    {
        aux[i++] = arr[l++];
    }
    while (r <= end)
    {
        aux[i++] = arr[r++];
    }

    for (i = start; i <= end; i++)
    {
        arr[i] = aux[i];
    }
}

inline mergeSortResult_t mergeSortError(hipError_t cudaStatus, char *msg)
{
    mergeSortResult_t error;
    error.cudaStatus = cudaStatus;
    error.msg = msg;
    return error;
}

inline mergeSortResult_t mergeSortSuccess()
{
    mergeSortResult_t success;
    success.cudaStatus = hipSuccess;
    return success;
}

inline mergeSortResult_t doMergeSortWithCuda(int *arr, unsigned int size, int *dev_arr, int *dev_aux)
{
    const unsigned int last = size - 1;
    const unsigned size = size * sizeof(int);
    unsigned int threadCount;
    hipError_t cudaStatus;
    char msg[1024];

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_arr, arr, size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        return mergeSortError(cudaStatus, "hipMemcpy failed!");
    }

    for (unsigned int blockSize = 2; blockSize < 2 * size; blockSize *= 2)
    {
        threadCount = size / blockSize;
        if (size % blockSize > 0)
        {
            threadCount++;
        }

        // Launch a kernel on the GPU with one thread for each block.
        mergeSortKernel<<<1, threadCount>>>(dev_arr, dev_aux, blockSize, last);

        // Check for any errors launching the kernel
        cudaStatus = hipGetLastError();
        if (cudaStatus != hipSuccess)
        {
            sprintf(msg, "mergeSortKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
            return mergeSortError(cudaStatus, msg);
        }

        // hipDeviceSynchronize waits for the kernel to finish, and returns
        // any errors encountered during the launch.
        cudaStatus = hipDeviceSynchronize();
        if (cudaStatus != hipSuccess)
        {
            sprintf(msg, "hipDeviceSynchronize returned error code %d after launching mergeSortKernel!\n", cudaStatus);
            return mergeSortError(cudaStatus, msg);
        }
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(arr, dev_arr, size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        return mergeSortError(cudaStatus, "hipMemcpy failed!");
    }

    return mergeSortSuccess();
}

hipError_t mergeSortWithCuda(int *arr, unsigned int size)
{
    const unsigned int size = size * sizeof(int);
    int *dev_arr = 0;
    int *dev_aux = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipSetDevice failed! Do you have a CUDA-capable GPU installed?");
        return cudaStatus;
    }

    // Allocate GPU buffers for two vectors (main and aux array).
    cudaStatus = hipMalloc((void **)&dev_arr, size);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        return cudaStatus;
    }

    cudaStatus = hipMalloc((void **)&dev_aux, size);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!");
        hipFree(dev_arr);
        return cudaStatus;
    }

    mergeSortResult_t result = doMergeSortWithCuda(arr, size, dev_arr, dev_aux);

    if (result.cudaStatus != hipSuccess)
    {
        fprintf(stderr, result.msg);
    }

    hipFree(dev_arr);
    hipFree(dev_aux);

    return cudaStatus;
}