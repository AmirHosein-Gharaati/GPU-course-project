#include "hip/hip_runtime.h"

#include <stdlib.h>

extern "C"
{
#include "cuda_sort.h"
}

__global__ void simple_mergesort(int *data, int *dataAux, int begin, int end, int depth)
{
    int middle = (end + begin) / 2;
    int i0 = begin;
    int i1 = middle;
    int index;
    int n = end - begin;

    hipStream_t s, s1;

    if (n < 2)
    {
        return;
    }

    hipStreamCreateWithFlags(&s, hipStreamNonBlocking);
    simple_mergesort<<<1, 1, 0, s>>>(data, dataAux, begin, middle, depth + 1);
    hipStreamDestroy(s);

    hipStreamCreateWithFlags(&s1, hipStreamNonBlocking);
    simple_mergesort<<<1, 1, 0, s1>>>(data, dataAux, middle, end, depth + 1);
    hipStreamDestroy(s1);

    hipDeviceSynchronize();

    for (index = begin; index < end; index++)
    {
        if (i0 < middle && (i1 >= end || data[i0] <= data[i1]))
        {
            dataAux[index] = data[i0];
            i0++;
        }
        else
        {
            dataAux[index] = data[i1];
            i1++;
        }
    }

    for (index = begin; index < end; index++)
    {
        data[index] = dataAux[index];
    }
}

extern "C" void gpu_merge_sort(int *array, int size)
{
    int *gpuData;
    int *gpuAuxData;
    int left = 0;
    int right = size;

    hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, MAX_DEPTH);

    hipMalloc((void **)&gpuData, size * sizeof(int));
    hipMalloc((void **)&gpuAuxData, size * sizeof(int));
    hipMemcpy(gpuData, array, size * sizeof(int), hipMemcpyHostToDevice);

    simple_mergesort<<<1, 1>>>(gpuData, gpuAuxData, left, right, 0);
    hipDeviceSynchronize();

    hipMemcpy(array, gpuData, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(gpuAuxData);
    hipFree(gpuData);

    hipDeviceReset();
}