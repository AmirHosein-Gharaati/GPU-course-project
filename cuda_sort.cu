#include "hip/hip_runtime.h"
#include <sys/time.h>
#include "cuda_sort.h"

void mergesort(long *data, long size, dim3 threadsPerBlock, dim3 blocksPerGrid)
{

    //
    // Allocate two arrays on the GPU
    // we switch back and forth between them during the sort
    //
    long *D_data;
    long *D_swp;
    dim3 *D_threads;
    dim3 *D_blocks;

    // Actually allocate the two arrays
    tm();
    checkCudaErrors(hipMalloc((void **)&D_data, size * sizeof(long)));
    checkCudaErrors(hipMalloc((void **)&D_swp, size * sizeof(long)));
    if (verbose)
        std::cout << "hipMalloc device lists: " << tm() << " microseconds\n";

    // Copy from our input list into the first array
    checkCudaErrors(hipMemcpy(D_data, data, size * sizeof(long), hipMemcpyHostToDevice));
    if (verbose)
        std::cout << "hipMemcpy list to device: " << tm() << " microseconds\n";

    //
    // Copy the thread / block info to the GPU as well
    //
    checkCudaErrors(hipMalloc((void **)&D_threads, sizeof(dim3)));
    checkCudaErrors(hipMalloc((void **)&D_blocks, sizeof(dim3)));

    if (verbose)
        std::cout << "hipMalloc device thread data: " << tm() << " microseconds\n";
    checkCudaErrors(hipMemcpy(D_threads, &threadsPerBlock, sizeof(dim3), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(D_blocks, &blocksPerGrid, sizeof(dim3), hipMemcpyHostToDevice));

    if (verbose)
        std::cout << "hipMemcpy thread data to device: " << tm() << " microseconds\n";

    long *A = D_data;
    long *B = D_swp;

    long nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
                    blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;

    //
    // Slice up the list and give pieces of it to each thread, letting the pieces grow
    // bigger and bigger until the whole list is sorted
    //
    for (int width = 2; width < (size << 1); width <<= 1)
    {
        long slices = size / ((nThreads)*width) + 1;

        if (verbose)
        {
            std::cout << "mergeSort - width: " << width
                      << ", slices: " << slices
                      << ", nThreads: " << nThreads << '\n';
            tm();
        }

        // Actually call the kernel
        gpu_mergesort<<<blocksPerGrid, threadsPerBlock>>>(A, B, size, width, slices, D_threads, D_blocks);

        if (verbose)
            std::cout << "call mergesort kernel: " << tm() << " microseconds\n";

        // Switch the input / output arrays instead of copying them around
        A = A == D_data ? D_swp : D_data;
        B = B == D_data ? D_swp : D_data;
    }

    //
    // Get the list back from the GPU
    //
    tm();
    checkCudaErrors(hipMemcpy(data, A, size * sizeof(long), hipMemcpyDeviceToHost));
    if (verbose)
        std::cout << "hipMemcpy list back to host: " << tm() << " microseconds\n";

    // Free the GPU memory
    checkCudaErrors(hipFree(A));
    checkCudaErrors(hipFree(B));
    if (verbose)
        std::cout << "hipFree: " << tm() << " microseconds\n";
}

// GPU helper function
// calculate the id of the current thread
__device__ unsigned int getIdx(dim3 *threads, dim3 *blocks)
{
    int x;
    return threadIdx.x +
           threadIdx.y * (x = threads->x) +
           threadIdx.z * (x *= threads->y) +
           blockIdx.x * (x *= threads->z) +
           blockIdx.y * (x *= blocks->z) +
           blockIdx.z * (x *= blocks->y);
}

//
// Perform a full mergesort on our section of the data.
//
__global__ void gpu_mergesort(long *source, long *dest, long size, long width, long slices, dim3 *threads, dim3 *blocks)
{
    unsigned int idx = getIdx(threads, blocks);
    long start = width * idx * slices,
         middle,
         end;

    for (long slice = 0; slice < slices; slice++)
    {
        if (start >= size)
            break;

        middle = min(start + (width >> 1), size);
        end = min(start + width, size);
        gpu_bottomUpMerge(source, dest, start, middle, end);
        start += width;
    }
}

//
// Finally, sort something
// gets called by gpu_mergesort() for each slice
//
__device__ void gpu_bottomUpMerge(long *source, long *dest, long start, long middle, long end)
{
    long i = start;
    long j = middle;
    for (long k = start; k < end; k++)
    {
        if (i < middle && (j >= end || source[i] < source[j]))
        {
            dest[k] = source[i];
            i++;
        }
        else
        {
            dest[k] = source[j];
            j++;
        }
    }
}

// read data into a minimal linked list
typedef struct
{
    int v;
    void *next;
} LinkNode;

// helper function for reading numbers from stdin
// it's 'optimized' not to check validity of the characters it reads in..
long readList(long **list)
{
    tm();
    long v, size = 0;
    LinkNode *node = 0;
    LinkNode *first = 0;
    while (std::cin >> v)
    {
        LinkNode *next = new LinkNode();
        next->v = v;
        if (node)
            node->next = next;
        else
            first = next;
        node = next;
        size++;
    }

    if (size)
    {
        *list = new long[size];
        LinkNode *node = first;
        long i = 0;
        while (node)
        {
            (*list)[i++] = node->v;
            node = (LinkNode *)node->next;
        }
    }

    if (verbose)
        std::cout << "read stdin: " << tm() << " microseconds\n";

    return size;
}

//
// Get the time (in microseconds) since the last call to tm();
// the first value returned by this must not be trusted
//
timeval tStart;
int tm()
{
    timeval tEnd;
    gettimeofday(&tEnd, 0);
    int t = (tEnd.tv_sec - tStart.tv_sec) * 1000000 + tEnd.tv_usec - tStart.tv_usec;
    tStart = tEnd;
    return t;
}