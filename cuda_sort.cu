
#include <stdlib.h>

extern "C"
{
#include "cuda_sort.h"
}

extern "C" void gpu_merge_sort(int *array, int size)
{
    int *gpuData;
    int *gpuAuxData;
    int left = 0;
    int right = size;

    hipDeviceSetLimit(cudaLimitDevRuntimeSyncDepth, MAX_DEPTH);

    hipMalloc((void **)&gpuData, size * sizeof(int));
    hipMalloc((void **)&gpuAuxData, size * sizeof(int));
    hipMemcpy(gpuData, array, size * sizeof(int), hipMemcpyHostToDevice);

    simple_mergesort<<<1, 1>>>(gpuData, gpuAuxData, left, right, 0);
    hipDeviceSynchronize();

    hipMemcpy(array, gpuData, size * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(gpuAuxData);
    hipFree(gpuData);

    hipDeviceReset();
}