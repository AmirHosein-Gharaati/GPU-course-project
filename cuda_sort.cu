#include "hip/hip_runtime.h"
#include <sys/time.h>
#include "cuda_sort.h"

__global__ void gpu_mergesort(int *source, int *dest, int size, int width, int slices, dim3 *threads, dim3 *blocks);
__device__ void gpu_bottomUpMerge(int *source, int *dest, int start, int middle, int end)

    void mergesort(int *data, int size)
{

    //
    // Allocate two arrays on the GPU
    // we switch back and forth between them during the sort
    //
    int *D_data;
    int *D_swp;
    dim3 *D_threads;
    dim3 *D_blocks;

    dim3 threadsPerBlock;
    dim3 blocksPerGrid;

    threadsPerBlock.x = 32;
    threadsPerBlock.y = 1;
    threadsPerBlock.z = 1;

    blocksPerGrid.x = 8;
    blocksPerGrid.y = 1;
    blocksPerGrid.z = 1;

    // Actually allocate the two arrays
    tm();
    (hipMalloc((void **)&D_data, size * sizeof(int)));
    (hipMalloc((void **)&D_swp, size * sizeof(int)));
    // if (verbose)
    //     std::cout << "hipMalloc device lists: " << tm() << " microseconds\n";

    // Copy from our input list into the first array
    (hipMemcpy(D_data, data, size * sizeof(int), hipMemcpyHostToDevice));
    // if (verbose)
    //     std::cout << "hipMemcpy list to device: " << tm() << " microseconds\n";

    //
    // Copy the thread / block info to the GPU as well
    //
    (hipMalloc((void **)&D_threads, sizeof(dim3)));
    (hipMalloc((void **)&D_blocks, sizeof(dim3)));

    // if (verbose)
    //     std::cout << "hipMalloc device thread data: " << tm() << " microseconds\n";
    (hipMemcpy(D_threads, &threadsPerBlock, sizeof(dim3), hipMemcpyHostToDevice));
    (hipMemcpy(D_blocks, &blocksPerGrid, sizeof(dim3), hipMemcpyHostToDevice));

    // if (verbose)
    //     std::cout << "hipMemcpy thread data to device: " << tm() << " microseconds\n";

    int *A = D_data;
    int *B = D_swp;

    int nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
                   blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;

    //
    // Slice up the list and give pieces of it to each thread, letting the pieces grow
    // bigger and bigger until the whole list is sorted
    //
    for (int width = 2; width < (size << 1); width <<= 1)
    {
        int slices = size / ((nThreads)*width) + 1;

        // if (verbose)
        // {
        //     std::cout << "mergeSort - width: " << width
        //               << ", slices: " << slices
        //               << ", nThreads: " << nThreads << '\n';
        //     tm();
        // }

        // Actually call the kernel
        gpu_mergesort<<<blocksPerGrid, threadsPerBlock>>>(A, B, size, width, slices, D_threads, D_blocks);

        // if (verbose)
        //     std::cout << "call mergesort kernel: " << tm() << " microseconds\n";

        // Switch the input / output arrays instead of copying them around
        A = A == D_data ? D_swp : D_data;
        B = B == D_data ? D_swp : D_data;
    }

    //
    // Get the list back from the GPU
    //
    tm();
    (hipMemcpy(data, A, size * sizeof(int), hipMemcpyDeviceToHost));
    // if (verbose)
    //     std::cout << "hipMemcpy list back to host: " << tm() << " microseconds\n";

    // Free the GPU memory
    (hipFree(A));
    (hipFree(B));
    // if (verbose)
    //     std::cout << "hipFree: " << tm() << " microseconds\n";
}

// GPU helper function
// calculate the id of the current thread
__device__ unsigned int getIdx(dim3 *threads, dim3 *blocks)
{
    int x;
    return threadIdx.x +
           threadIdx.y * (x = threads->x) +
           threadIdx.z * (x *= threads->y) +
           blockIdx.x * (x *= threads->z) +
           blockIdx.y * (x *= blocks->z) +
           blockIdx.z * (x *= blocks->y);
}

//
// Perform a full mergesort on our section of the data.
//
__global__ void gpu_mergesort(int *source, int *dest, int size, int width, int slices, dim3 *threads, dim3 *blocks)
{
    unsigned int idx = getIdx(threads, blocks);
    int start = width * idx * slices,
        middle,
        end;

    for (int slice = 0; slice < slices; slice++)
    {
        if (start >= size)
            break;

        middle = min(start + (width >> 1), size);
        end = min(start + width, size);
        gpu_bottomUpMerge(source, dest, start, middle, end);
        start += width;
    }
}

//
// Finally, sort something
// gets called by gpu_mergesort() for each slice
//
__device__ void gpu_bottomUpMerge(int *source, int *dest, int start, int middle, int end)
{
    int i = start;
    int j = middle;
    for (int k = start; k < end; k++)
    {
        if (i < middle && (j >= end || source[i] < source[j]))
        {
            dest[k] = source[i];
            i++;
        }
        else
        {
            dest[k] = source[j];
            j++;
        }
    }
}

//
// Get the time (in microseconds) since the last call to tm();
// the first value returned by this must not be trusted
//
timeval tStart;
int tm()
{
    timeval tEnd;
    gettimeofday(&tEnd, 0);
    int t = (tEnd.tv_sec - tStart.tv_sec) * 1000000 + tEnd.tv_usec - tStart.tv_usec;
    tStart = tEnd;
    return t;
}